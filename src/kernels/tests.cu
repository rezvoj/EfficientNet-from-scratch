#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <numeric>
#include <string>
#include "Convolution.cuh"
#include "../utils/Exceptions.cuh"
#include "../utils/Math.cuh"



template <int FILTER_SIZE, int STRIDE>
void referenceDepthwiseConv(
        float* outTensor,
        const float* inTensor,
        const float* inFilters,
        const int BSize,
        const int CSize,
        const int inHSize,
        const int inWSize,
        const int outHSize, 
        const int outWSize) {
    // Calculate padding and tensor strides
    const int padHIdx = (FILTER_SIZE - 1) / 2;
    const int padWIdx = (FILTER_SIZE - 1) / 2;
    const int inStrideBIdx = CSize * inHSize * inWSize;
    const int inStrideCIdx = inHSize * inWSize;
    const int outStrideBIdx = CSize * outHSize * outWSize;
    const int outStrideCIdx = outHSize * outWSize;
    const int filterStrideCIdx = FILTER_SIZE * FILTER_SIZE;
    // Iterate over each output element to compute its value
    for (int batchIdx = 0; batchIdx < BSize; ++batchIdx) {
        for (int channelIdx = 0; channelIdx < CSize; ++channelIdx) {
            for (int outHIdx = 0; outHIdx < outHSize; ++outHIdx) {
                for (int outWIdx = 0; outWIdx < outWSize; ++outWIdx) {
                    // Compute the convolution sum for one output element
                    float sum = 0.0f;
                    for (int filterHIdx = 0; filterHIdx < FILTER_SIZE; ++filterHIdx) {
                        for (int filterWIdx = 0; filterWIdx < FILTER_SIZE; ++filterWIdx) {
                            // Map output coordinates to input coordinates
                            const int inHIdx = outHIdx * STRIDE + filterHIdx - padHIdx;
                            const int inWIdx = outWIdx * STRIDE + filterWIdx - padWIdx;
                            // Accumulate sum, applying zero-padding implicitly
                            if (inHIdx >= 0 && inHIdx < inHSize && inWIdx >= 0 && inWIdx < inWSize) {
                                const int inTensorIdx = batchIdx * inStrideBIdx 
                                    + channelIdx * inStrideCIdx 
                                    + inHIdx * inWSize + inWIdx;
                                const int filterIdx = channelIdx * filterStrideCIdx 
                                    + filterHIdx * FILTER_SIZE + filterWIdx;
                                sum += inTensor[inTensorIdx] * inFilters[filterIdx];
                            }
                        }
                    }
                    // Store the final computed value
                    const int outTensorIdx = batchIdx * outStrideBIdx 
                        + channelIdx * outStrideCIdx 
                        + outHIdx * outWSize + outWIdx;
                    outTensor[outTensorIdx] = sum;
                }
            }
        }
    }
}



template <int FILTER_SIZE, int STRIDE>
void referenceDepthwiseConvBackward(
        float* inputGradTensor,
        const float* outputGradTensor,
        const float* filtersTensor,
        const int BSize,
        const int CSize,
        const int inHSize,
        const int inWSize,
        const int outHSize,
        const int outWSize) {
    // Calculate padding and tensor strides
    const int padHIdx = (FILTER_SIZE - 1) / 2;
    const int padWIdx = (FILTER_SIZE - 1) / 2;
    const int inputGradStrideBIdx = CSize * inHSize * inWSize;
    const int inputGradStrideCIdx = inHSize * inWSize;
    const int outputGradStrideBIdx = CSize * outHSize * outWSize;
    const int outputGradStrideCIdx = outHSize * outWSize;
    const int filterStrideCIdx = FILTER_SIZE * FILTER_SIZE;
    // Zero-out the input gradient tensor before accumulation
    for (int i = 0; i < BSize * CSize * inHSize * inWSize; ++i) {
        inputGradTensor[i] = 0.0f;
    }
    // Iterate over each output gradient to scatter its contribution
    for (int batchIdx = 0; batchIdx < BSize; ++batchIdx) {
        for (int channelIdx = 0; channelIdx < CSize; ++channelIdx) {
            for (int outHIdx = 0; outHIdx < outHSize; ++outHIdx) {
                for (int outWIdx = 0; outWIdx < outWSize; ++outWIdx) {
                    // Get the current output gradient value
                    const int outputGradIdx = batchIdx * outputGradStrideBIdx
                        + channelIdx * outputGradStrideCIdx
                        + outHIdx * outWSize + outWIdx;
                    const float outputGradVal = outputGradTensor[outputGradIdx];
                    // Apply the filter to scatter the gradient
                    for (int filterHIdx = 0; filterHIdx < FILTER_SIZE; ++filterHIdx) {
                        for (int filterWIdx = 0; filterWIdx < FILTER_SIZE; ++filterWIdx) {
                            // Map output coordinates to input coordinates
                            const int inHIdx = outHIdx * STRIDE + filterHIdx - padHIdx;
                            const int inWIdx = outWIdx * STRIDE + filterWIdx - padWIdx;
                            // Accumulate gradient, applying zero-padding implicitly
                            if (inHIdx >= 0 && inHIdx < inHSize && inWIdx >= 0 && inWIdx < inWSize) {
                                const int filterIdx = channelIdx * filterStrideCIdx
                                    + filterHIdx * FILTER_SIZE + filterWIdx;
                                const float filterVal = filtersTensor[filterIdx];
                                const int inputGradIdx = batchIdx * inputGradStrideBIdx 
                                    + channelIdx * inputGradStrideCIdx
                                    + inHIdx * inWSize + inWIdx;
                                inputGradTensor[inputGradIdx] += outputGradVal * filterVal;
                            }
                        }
                    }
                }
            }
        }
    }
}



template <int FILTER_SIZE, int STRIDE>
void referenceDepthwiseConvBackwardGrad(
        float* filterGradTensor,
        const float* outputGradTensor,
        const float* inTensor,
        const int BSize,
        const int CSize,
        const int inHSize,
        const int inWSize,
        const int outHSize,
        const int outWSize) {
    // Calculate padding and tensor strides
    const int padHIdx = (FILTER_SIZE - 1) / 2;
    const int padWIdx = (FILTER_SIZE - 1) / 2;
    const int inStrideBIdx = CSize * inHSize * inWSize;
    const int inStrideCIdx = inHSize * inWSize;
    const int outputGradStrideBIdx = CSize * outHSize * outWSize;
    const int outputGradStrideCIdx = outHSize * outWSize;
    const int filterGradStrideCIdx = FILTER_SIZE * FILTER_SIZE;
    // Zero-out the filter gradient tensor before accumulation
    for (int i = 0; i < CSize * FILTER_SIZE * FILTER_SIZE; ++i) {
        filterGradTensor[i] = 0.0f;
    }
    // Iterate over each filter weight to compute its gradient
    for (int channelIdx = 0; channelIdx < CSize; ++channelIdx) {
        for (int filterHIdx = 0; filterHIdx < FILTER_SIZE; ++filterHIdx) {
            for (int filterWIdx = 0; filterWIdx < FILTER_SIZE; ++filterWIdx) {
                // Sum the gradient contribution from all batches and positions
                float gradientSum = 0.0f;
                for (int batchIdx = 0; batchIdx < BSize; ++batchIdx) {
                    for (int outHIdx = 0; outHIdx < outHSize; ++outHIdx) {
                        for (int outWIdx = 0; outWIdx < outWSize; ++outWIdx) {
                            // Map output coordinates to input coordinates
                            const int inHIdx = outHIdx * STRIDE + filterHIdx - padHIdx;
                            const int inWIdx = outWIdx * STRIDE + filterWIdx - padWIdx;
                            // Accumulate gradient, applying zero-padding implicitly
                            if (inHIdx >= 0 && inHIdx < inHSize && inWIdx >= 0 && inWIdx < inWSize) {
                                const int inTensorIdx = batchIdx * inStrideBIdx
                                    + channelIdx * inStrideCIdx
                                    + inHIdx * inWSize + inWIdx;
                                const float inputValue = inTensor[inTensorIdx];
                                const int outputGradTensorIdx = batchIdx * outputGradStrideBIdx
                                    + channelIdx * outputGradStrideCIdx
                                    + outHIdx * outWSize + outWIdx;
                                const float outputGradValue = outputGradTensor[outputGradTensorIdx];
                                gradientSum += inputValue * outputGradValue;
                            }
                        }
                    }
                }
                // Store the final computed gradient for the filter weight
                const int filterGradTensorIdx = channelIdx * filterGradStrideCIdx
                    + filterHIdx * FILTER_SIZE + filterWIdx;
                filterGradTensor[filterGradTensorIdx] = gradientSum;
            }
        }
    }
}



struct TestParams {
    int BSize;
    int CSize;
    int inHSize;
    int inWSize;
    std::string description;
};



template <int FILTER_SIZE, int STRIDE>
bool depthwiseConvTest(const TestParams& params) {
    // Extract test parameters from input struct
    const int BSize = params.BSize;
    const int CSize = params.CSize;
    const int inHSize = params.inHSize;
    const int inWSize = params.inWSize;
    // Define CUDA block dimensions
    constexpr int BLOCK_X_SIZE = 16;
    constexpr int BLOCK_Y_SIZE = 16;
    // Print test configuration details
    std::cout << "----------------------------------------" << std::endl;
    std::cout << "Description: " << params.description << std::endl;
    std::cout << "Filter: " << FILTER_SIZE << ", Stride: " << STRIDE;
    std::cout << ", Input: [" << BSize << "," << CSize << ",";
    std::cout << inHSize << "," << inWSize << "]" << std::endl;
    // Calculate output dimensions and tensor sizes
    const int outHSize = ceilDiv(inHSize, STRIDE);
    const int outWSize = ceilDiv(inWSize, STRIDE);
    const int inTensorSize = BSize * CSize * inHSize * inWSize;
    const int outTensorSize = BSize * CSize * outHSize * outWSize;
    const int filtersSize = CSize * FILTER_SIZE * FILTER_SIZE;
    // Allocate and initialize host memory with random values
    float* h_inTensor = new float[inTensorSize];
    float* h_inFilters = new float[filtersSize];
    float* h_outTensorCPU = new float[outTensorSize];
    float* h_outTensorGPU = new float[outTensorSize];
    for (int i = 0; i < inTensorSize; ++i) h_inTensor[i] = static_cast<float>(rand() % 10);
    for (int i = 0; i < filtersSize; ++i) h_inFilters[i] = static_cast<float>(rand() % 5 - 2);
    // Allocate and copy data to device memory
    float *d_inTensor, *d_inFilters, *d_outTensor;
    checkCuda(hipMalloc(&d_inTensor, inTensorSize * sizeof(float)));
    checkCuda(hipMalloc(&d_inFilters, filtersSize * sizeof(float)));
    checkCuda(hipMalloc(&d_outTensor, outTensorSize * sizeof(float)));
    checkCuda(hipMemcpy(d_inTensor, h_inTensor, inTensorSize * sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_inFilters, h_inFilters, filtersSize * sizeof(float), hipMemcpyHostToDevice));
    // Configure CUDA kernel launch parameters
    const int outHBlocks = ceilDiv(outHSize, BLOCK_Y_SIZE);
    const dim3 gridDim(ceilDiv(outWSize, BLOCK_X_SIZE), BSize * CSize * outHBlocks);
    const dim3 blockDim(BLOCK_X_SIZE, BLOCK_Y_SIZE);
    // Launch CUDA kernel and copy back to host
    depthwiseConvForward<BLOCK_X_SIZE, BLOCK_Y_SIZE, FILTER_SIZE, STRIDE><<<gridDim, blockDim>>>(
        d_outTensor, d_inTensor, d_inFilters, 
        CSize, outHSize, outWSize, outHBlocks, 
        inHSize, inWSize
    ); 
    checkCudaLastError();
    checkCuda(hipMemcpy(h_outTensorGPU, d_outTensor, outTensorSize * sizeof(float), hipMemcpyDeviceToHost));
    // Run reference dethwise convolution forward implementation
    referenceDepthwiseConv<FILTER_SIZE, STRIDE>(
        h_outTensorCPU,
        h_inTensor, h_inFilters,
        BSize, CSize,
        inHSize, inWSize,
        outHSize, outWSize
    );
    // Validate results by comparing reference CPU and GPU outputs
    bool success = true;
    for (int i = 0; i < outTensorSize; ++i) {
        if (std::abs(h_outTensorCPU[i] - h_outTensorGPU[i]) > 1e-5) {
            std::cerr << "  [FAIL] Mismatch at index " << i;
            std::cerr << "! CPU: " << h_outTensorCPU[i];
            std::cerr << ", GPU: " << h_outTensorGPU[i] << std::endl;
            success = false;
            break;
        }
    }
    // Print test result and cleanup resources
    if (success) {
        std::cout << "[PASS]" << std::endl;
    }
    delete[] h_inTensor;
    delete[] h_inFilters;
    delete[] h_outTensorCPU;
    delete[] h_outTensorGPU;
    checkCuda(hipFree(d_inTensor));
    checkCuda(hipFree(d_inFilters));
    checkCuda(hipFree(d_outTensor));
    return success;
}



template <int FILTER_SIZE, int STRIDE>
bool depthwiseConvBackwardTest(const TestParams& params) {
    // Extract test parameters from input struct
    const int BSize = params.BSize;
    const int CSize = params.CSize;
    const int inHSize = params.inHSize;
    const int inWSize = params.inWSize;
    // Define CUDA block dimensions
    constexpr int BLOCK_X_SIZE = 16;
    constexpr int BLOCK_Y_SIZE = 16;
    // Print test configuration details
    std::cout << "----------------------------------------" << std::endl;
    std::cout << "Description: " << params.description << std::endl;
    std::cout << "Filter: " << FILTER_SIZE << ", Stride: " << STRIDE;
    std::cout << ", Input Grad Size: [" << BSize << "," << CSize << ",";
    std::cout << inHSize << "," << inWSize << "]" << std::endl;
    // Calculate output dimensions and tensor sizes
    const int outHSize = ceilDiv(inHSize, STRIDE);
    const int outWSize = ceilDiv(inWSize, STRIDE);
    const int inputGradTensorSize = BSize * CSize * inHSize * inWSize;
    const int outputGradTensorSize = BSize * CSize * outHSize * outWSize;
    const int filtersTensorSize = CSize * FILTER_SIZE * FILTER_SIZE;
    // Allocate and initialize host memory with random values
    float* h_outputGradTensor = new float[outputGradTensorSize];
    float* h_filtersTensor = new float[filtersTensorSize];
    float* h_inputGradTensor_CPU = new float[inputGradTensorSize];
    float* h_inputGradTensor_GPU = new float[inputGradTensorSize];
    for (int i = 0; i < outputGradTensorSize; ++i) h_outputGradTensor[i] = static_cast<float>(rand() % 10 - 5);
    for (int i = 0; i < filtersTensorSize; ++i) h_filtersTensor[i] = static_cast<float>(rand() % 5 - 2);
    // Allocate and copy data to device memory
    float *d_outputGradTensor, *d_filtersTensor, *d_inputGradTensor;
    checkCuda(hipMalloc(&d_outputGradTensor, outputGradTensorSize * sizeof(float)));
    checkCuda(hipMalloc(&d_filtersTensor, filtersTensorSize * sizeof(float)));
    checkCuda(hipMalloc(&d_inputGradTensor, inputGradTensorSize * sizeof(float)));
    checkCuda(hipMemcpy(d_outputGradTensor, h_outputGradTensor, outputGradTensorSize * sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_filtersTensor, h_filtersTensor, filtersTensorSize * sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemset(d_inputGradTensor, 0, inputGradTensorSize * sizeof(float)));
    // Configure CUDA kernel launch parameters
    const int inHBlocks = ceilDiv(inHSize, BLOCK_Y_SIZE);
    const dim3 gridDim(ceilDiv(inWSize, BLOCK_X_SIZE), BSize * CSize * inHBlocks);
    const dim3 blockDim(BLOCK_X_SIZE, BLOCK_Y_SIZE);
    // Launch CUDA kernel and copy back to host
    depthwiseConvBackward<BLOCK_X_SIZE, BLOCK_Y_SIZE, FILTER_SIZE, STRIDE><<<gridDim, blockDim>>>(
        d_inputGradTensor, d_outputGradTensor, d_filtersTensor, 
        CSize, inHSize, inWSize, 
        inHBlocks, outHSize, outWSize
    );
    checkCudaLastError();
    checkCuda(hipMemcpy(h_inputGradTensor_GPU, d_inputGradTensor, inputGradTensorSize * sizeof(float), hipMemcpyDeviceToHost));
    // Run reference depthwise convolution backward implementation
    referenceDepthwiseConvBackward<FILTER_SIZE, STRIDE>(
        h_inputGradTensor_CPU, h_outputGradTensor, h_filtersTensor,
        BSize, CSize,
        inHSize, inWSize,
        outHSize, outWSize
    );
    // Validate results by comparing reference CPU and GPU outputs
    bool success = true;
    for (int i = 0; i < inputGradTensorSize; ++i) {
        if (std::abs(h_inputGradTensor_CPU[i] - h_inputGradTensor_GPU[i]) > 1e-4) {
            std::cerr << "  [FAIL] Mismatch at index " << i;
            std::cerr << "! CPU: " << h_inputGradTensor_CPU[i];
            std::cerr << ", GPU: " << h_inputGradTensor_GPU[i] << std::endl;
            success = false;
            break;
        }
    }
    // Print test result and cleanup resources
    if (success) {
        std::cout << "[PASS]" << std::endl;
    }
    delete[] h_outputGradTensor;
    delete[] h_filtersTensor;
    delete[] h_inputGradTensor_CPU;
    delete[] h_inputGradTensor_GPU;
    checkCuda(hipFree(d_outputGradTensor));
    checkCuda(hipFree(d_filtersTensor));
    checkCuda(hipFree(d_inputGradTensor));
    return success;
}



template <int FILTER_SIZE, int STRIDE>
bool depthwiseConvBackwardGradTest(const TestParams& params) {
    // Extract test parameters from the input struct
    const int BSize = params.BSize;
    const int CSize = params.CSize;
    const int inHSize = params.inHSize;
    const int inWSize = params.inWSize;
    // Define a common 1D block size for the CUDA kernel
    constexpr int BLOCK_SIZE = 256;
    // Print test configuration details
    std::cout << "----------------------------------------" << std::endl;
    std::cout << "Description: " << params.description << std::endl;
    std::cout << "Filter: " << FILTER_SIZE << ", Stride: " << STRIDE;
    std::cout << ", Input Size: [" << BSize << "," << CSize << "," << inHSize << "," << inWSize << "]" << std::endl;
    // Calculate output dimensions and tensor sizes
    const int outHSize = ceilDiv(inHSize, STRIDE);
    const int outWSize = ceilDiv(inWSize, STRIDE);
    const int inTensorSize = BSize * CSize * inHSize * inWSize;
    const int outputGradTensorSize = BSize * CSize * outHSize * outWSize;
    const int filterGradTensorSize = CSize * FILTER_SIZE * FILTER_SIZE;
    // Allocate and initialize host memory with random values
    float* h_inTensor = new float[inTensorSize];
    float* h_outputGradTensor = new float[outputGradTensorSize];
    float* h_filterGradTensor_CPU = new float[filterGradTensorSize];
    float* h_filterGradTensor_GPU = new float[filterGradTensorSize];
    for (int i = 0; i < inTensorSize; ++i) h_inTensor[i] = static_cast<float>(rand() % 10 - 5) / 5.0f;
    for (int i = 0; i < outputGradTensorSize; ++i) h_outputGradTensor[i] = static_cast<float>(rand() % 10 - 5) / 5.0f;
    // Allocate and copy data to device memory
    float *d_inTensor, *d_outputGradTensor, *d_filterGradTensor;
    checkCuda(hipMalloc(&d_inTensor, inTensorSize * sizeof(float)));
    checkCuda(hipMalloc(&d_outputGradTensor, outputGradTensorSize * sizeof(float)));
    checkCuda(hipMalloc(&d_filterGradTensor, filterGradTensorSize * sizeof(float)));
    checkCuda(hipMemcpy(d_inTensor, h_inTensor, inTensorSize * sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_outputGradTensor, h_outputGradTensor, outputGradTensorSize * sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemset(d_filterGradTensor, 0, filterGradTensorSize * sizeof(float)));
    // Configure and launch the CUDA kernel and results from device to host
    const dim3 gridDim(CSize * FILTER_SIZE * FILTER_SIZE);
    const dim3 blockDim(BLOCK_SIZE);
    depthwiseConvBackwardGrad<BLOCK_SIZE, FILTER_SIZE, STRIDE><<<gridDim, blockDim>>>(
        d_filterGradTensor, d_outputGradTensor, d_inTensor,
        BSize, CSize, outHSize * outWSize, outWSize, 
        inHSize, inWSize
    );
    checkCudaLastError();
    checkCuda(hipMemcpy(h_filterGradTensor_GPU, d_filterGradTensor, filterGradTensorSize * sizeof(float), hipMemcpyDeviceToHost));
    // Run the reference implementation on the CPU to get the expected result
    referenceDepthwiseConvBackwardGrad<FILTER_SIZE, STRIDE>(
        h_filterGradTensor_CPU, h_outputGradTensor, h_inTensor,
        BSize, CSize,
        inHSize, inWSize,
        outHSize, outWSize
    );
    // Validate the results by comparing the GPU output against the CPU reference
    bool success = true;
    for (int i = 0; i < filterGradTensorSize; ++i) {
        if (std::abs(h_filterGradTensor_CPU[i] - h_filterGradTensor_GPU[i]) > 1e-3) {
            std::cerr << "  [FAIL] Mismatch at index " << i;
            std::cerr << "! CPU: " << h_filterGradTensor_CPU[i];
            std::cerr << ", GPU: " << h_filterGradTensor_GPU[i] << std::endl;
            success = false;
            break;
        }
    }
    // Print the final test result and clean up allocated resources
    if (success) {
        std::cout << "[PASS]" << std::endl;
    }
    delete[] h_inTensor;
    delete[] h_outputGradTensor;
    delete[] h_filterGradTensor_CPU;
    delete[] h_filterGradTensor_GPU;
    checkCuda(hipFree(d_inTensor));
    checkCuda(hipFree(d_outputGradTensor));
    checkCuda(hipFree(d_filterGradTensor));
    return success;
}



int main() {
    // Seed the random number generator for test data and initialize counters
    srand(static_cast<unsigned int>(time(0)));
    int passed_count = 0;
    int failed_count = 0;
    // Define a set of common test parameters to be used across all test cases
    std::vector<TestParams> test_cases = {
        {2, 4, 41, 35, "Basic non-aligned dimensions"},
        {1, 1, 8, 8, "Single item, small dimensions"},
        {4, 8, 32, 64, "Dimensions potentially aligned with block sizes"},
        {3, 6, 50, 50, "Larger non-aligned dimensions"}
    };
    // Test the forward depthwise convolution
    std::cout << "=========== DEPTHWISE CONV FORWARD TESTS ===========" << std::endl;
    for (const auto& params : test_cases) {
        depthwiseConvTest<3, 1>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvTest<3, 2>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvTest<5, 1>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvTest<5, 3>(params) ? passed_count++ : failed_count++;
    }
    // Test the backward pass for the input gradient
    std::cout << "\n====== DEPTHWISE CONV BACKWARD INPUT GRADIENT TESTS ======" << std::endl;
    for (const auto& params : test_cases) {
        depthwiseConvBackwardTest<3, 1>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvBackwardTest<3, 2>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvBackwardTest<5, 1>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvBackwardTest<5, 3>(params) ? passed_count++ : failed_count++;
    }
    // Test the backward pass for the filter gradient
    std::cout << "\n===== DEPTHWISE CONV BACKWARD FILTER GRADIENT TESTS =====" << std::endl;
    for (const auto& params : test_cases) {
        depthwiseConvBackwardGradTest<3, 1>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvBackwardGradTest<3, 2>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvBackwardGradTest<5, 1>(params) ? passed_count++ : failed_count++;
    }
    for (const auto& params : test_cases) {
        depthwiseConvBackwardGradTest<5, 3>(params) ? passed_count++ : failed_count++;
    }
    // Print the final summary of all test results
    std::cout << "\n============== TEST SUMMARY ===============" << std::endl;
    std::cout << "Total test cases ran: " << (passed_count + failed_count) << std::endl;
    std::cout << "Passed: " << passed_count << std::endl;
    std::cout << "Failed: " << failed_count << std::endl;
    std::cout << "===========================================" << std::endl;
    return 0;
}
