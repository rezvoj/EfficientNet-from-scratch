// src/utils/CudaUtils.cu

#include "CudaUtils.hpp"
#include <cstdio>
#include <cstdlib>

// Definition for general CUDA errors
void gpuAssert(hipError_t code, const char *file, int line, bool abort) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// --- NEW: Definitions for cuBLAS errors ---

// Helper to convert hipblasStatus_t to string
const char* cublasGetErrorString(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
    }
    return "UNKNOWN_CUBLAS_ERROR";
}

void cublasAssert(hipblasStatus_t status, const char *file, int line, bool abort) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLASassert: %s %s %d\n", cublasGetErrorString(status), file, line);
        // FIX: Use the 'status' variable, not 'code'.
        if (abort) exit(status);
    }
}